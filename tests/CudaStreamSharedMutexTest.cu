#include "hip/hip_runtime.h"
/*
Copyright [2024] [Yao Yao]

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.

*/

#include "../cuda_utils.h"
#include <gtest/gtest.h>
#include <thread>
#include <chrono>
#include "CudaStreamSharedMutex.h"
#include <random>

__global__ static void kernelWrite(volatile uint32_t* data)
{
    uint32_t ref = *data;
    for (uint32_t i = 0; i < 5000U; i++) {
        ref = i;
        *data = ref;
        kassert(ref == *data);
    }
}
__global__ static void kernelRead(const volatile uint32_t* data)
{
    const uint32_t ref = *data;
    for (uint32_t i = 0; i < 5000U; i++) {
        kassert(ref == *data);
    }
}

TEST(CudaStreamSharedMutexTest, random)
{
    using namespace cudapp;
    const auto evPoolHolder = createPooledCudaEvent();

    CudaStreamSharedMutex sharedMutex;
    const auto mem = allocCudaMem<uint32_t, CudaMemType::kDevice>(1);
    cudaCheck(hipMemset(mem.get(), 0, sizeof(*mem.get())));

    auto task = [&mem, &sharedMutex](){
        const CudaStream stream = makeCudaStream();
        std::default_random_engine rng{std::random_device{}()};
        std::bernoulli_distribution dist{0.1f};
        for (int i = 0; i < 1000; i++) {
            if (dist(rng)) {
                auto lk = sharedMutex.acquire(stream.get());
                kernelWrite<<<1, 1, 0, stream.get()>>>(mem.get());
                cudaCheck(hipGetLastError());
            }
            else {
                auto lk = sharedMutex.acquireShared(stream.get());
                kernelRead<<<1, 1, 0, stream.get()>>>(mem.get());
                cudaCheck(hipGetLastError());
            }
        }
        cudaCheck(hipStreamSynchronize(stream.get()));
    };
    std::vector<std::thread> threads;
    const int nbThrds = 4;

    threads.reserve(nbThrds);
    for (int i = 0; i < nbThrds; i++) {
        threads.emplace_back(task);
    }
    for (int i = 0; i < nbThrds; i++) {
        EXPECT_TRUE(threads.at(i).joinable());
        threads.at(i).join();
    }
}
