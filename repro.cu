#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(hipTextureObject_t texture)
{
    const float val = tex2D<float>(texture, 2.1f, 3.2f);
    printf("val = %f\n", val);
}

hipError_t launchKernel(hipTextureObject_t tex, hipStream_t stream)
{
    kernel<<<1, 1, 0, stream>>>(tex);
    return hipGetLastError();
}
